#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o 1_a_crack 1_a_crack.cu
    ./1_a_crack
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "AA1111";
	char plain_password2[] = "BB2222";
	char plain_password3[] = "SS3333";
	char plain_password4[] = "ZZ4444";


	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;
	char *p1 = plain_password1;
	char *p2 = plain_password2;
	char *p3 = plain_password3;
	char *p4 = plain_password4;

	while(*a == *p1) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		a++;
		p1++;
	}
	
	while(*b == *p2) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		b++;
		p2++;
	}

	while(*c == *p3) { 
		if(*c == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		c++;
		p3++;
	}

	while(*d == *p4) { 
		if(*d == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		d++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char a1,b2,c3,d4;//variables 

	char password[7];
	password[6] = '\0';
	//block id threrad id initilized
	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(a1='0'; a1<='9'; a1++){
		for(b2='0'; b2<='9'; b2++){
			for(c3='0'; c3<='9'; c3++){
				for(d4='0'; d4<='9'; d4++){
					password[2] = a1;
					password[3] = b2;
					password[4] = c3;
					password[5] = d4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}
//time difference
int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);
	//kernal function that determine block and threads to use
	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}
